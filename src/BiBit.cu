/*
 ============================================================================
 Name        : BiBit.cu
 Author      : Aurelio Lopez-Fernandez
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <thread>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <stdint.h>
#include <sys/time.h>
#include <unistd.h>
#include <inttypes.h>
#include <iterator>
#include <utility>
#include <algorithm>
#include <string>
#include <sstream>
#include <fstream>
#include <cstdlib>
#include <time.h>
#include <set>
#include <vector>
#include <map>
#include <unordered_set>
#include <mutex>
using namespace std;

namespace {
template<typename T>
std::size_t make_hash(const T& v) {
	return std::hash<T>()(v);
}

void hash_combine(std::size_t& h, const std::size_t& v) {
	h ^= v + 0x9e3779b9 + (h << 6) + (h >> 2);
}

template<typename T>
struct hash_container {
	size_t operator()(const T& v) const {
		size_t h = 0;
		for (const auto& e : v) {
			hash_combine(h, make_hash(e));
		}
		return h;
	}
};
}

namespace std {
template<typename T, typename U>
struct hash<pair<T, U>> {
	size_t operator()(const pair<T, U>& v) const {
		size_t h = make_hash(v.first);
		hash_combine(h, make_hash(v.second));
		return h;
	}
};

template<typename ... T>
struct hash<vector<T...>> : hash_container<vector<T...>> {
};

template<typename ... T>
struct hash<map<T...>> : hash_container<map<T...>> {
};
}

char *inputFile; // Input matrix.
ulong cCols, cColsTotal, cRows, cRowsPerThread, patternSize; // Pattern size: We will work with 32 bit or 64 bit in the encoding phase.
int cMnr, cMnc; // minimum number of rows allowed in a valid bicluster and minimum number of columns allowed in a valid bicluster
int output, deviceCount;
__constant__ ulong rowsPerThread; // 8 bytes
__constant__ ulong cols; // 8 bytes
__constant__ ulong numThreads; // 8 bytes
__constant__ ulong rows; // 8 bytes
__constant__ int mnr; // 4 bytes
__constant__ int mnc; // 4 bytes
__device__ unsigned long long int totalBiclusters;
__device__ unsigned long long int numPatFiltered;
ulong maxPatterns;
ulong maxThreadsPerBlock;
ulong maxBlocksPerGrid;
ulong maxIteratorGPU;
ulong lastBlocksGrid;
long long totales;
std::unordered_set<std::vector<uint64_t>> setPatterns64;
std::unordered_set<std::vector<uint32_t>> setPatterns32;
uint64_t *aResultColsCpu;
long long *aPatFilteredCpu;

__global__ void getPatterns(ulong maxPatterns, uint64_t *aResultCols, int id,
		ulong bicsPerGpuPrevious, uint64_t *mInputData, long long *patFiltered,
		ulong totalPatterns, ulong patternsPerRun, int iter, ulong totalFor) {
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
	ulong pattern = idTh + (totalFor * (iter - 1)) + (id * bicsPerGpuPrevious)
			+ totalPatterns;
	if (idTh < patternsPerRun && pattern < maxPatterns) {
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0) {
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--) {
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0) {
				r2 = (j + auxPat) + (r1 + 1);
			}
		}

		if (r1 < rows && r2 < rows) {
			ulong totalOnes = 0;
			for (ulong j = 0; j < cols; j++) {
				uint64_t rAnd = *(mInputData + r1 * cols + j)
						& *(mInputData + r2 * cols + j);
				*(aResultCols + idTh * cols + j) = rAnd;
				while (rAnd) {
					if (rAnd & 1 == 1) {
						totalOnes++;
					}
					rAnd >>= 1;
				}
			}
			if (totalOnes >= mnc) {
				unsigned long long int current_val = atomicAdd(&numPatFiltered,
						1);
				*(patFiltered + current_val) = idTh;
			}
		}
	}
}

__global__ void getPatterns(ulong maxPatterns, uint32_t *aResultCols, int id,
		ulong bicsPerGpuPrevious, uint32_t *mInputData, long long *patFiltered,
		ulong totalPatterns, ulong patternsPerRun, int iter, ulong totalFor) {
	ulong idTh = blockIdx.x * blockDim.x + threadIdx.x;
	ulong pattern = idTh + (totalFor * (iter - 1)) + (id * bicsPerGpuPrevious)
			+ totalPatterns;
	if (idTh < patternsPerRun && pattern < maxPatterns) {
		long r1 = 0;
		long r2 = -1;
		long auxPat = pattern - rows + 1;
		if (auxPat < 0) {
			r2 = auxPat + rows;
		}
		for (ulong j = rows - 2; r2 == -1; j--) {
			auxPat = auxPat - j;
			r1++;
			if (auxPat < 0) {
				r2 = (j + auxPat) + (r1 + 1);
			}
		}
		if (r1 < rows && r2 < rows) {
			ulong totalOnes = 0;
			for (ulong j = 0; j < cols; j++) {
				uint32_t rAnd = *(mInputData + r1 * cols + j)
						& *(mInputData + r2 * cols + j);
				*(aResultCols + idTh * cols + j) = rAnd;
				while (rAnd) {
					if (rAnd & 1 == 1) {
						totalOnes++;
					}
					rAnd >>= 1;
				}
			}
			if (totalOnes >= mnc) {
				unsigned long long int current_val = atomicAdd(&numPatFiltered,
						1);
				*(patFiltered + current_val) = idTh;
			}
		}
	}
}

__global__ void generateBiclusters(uint64_t *aResultCols, int id,
		uint64_t *mInputData, long long *patFiltered, uint8_t *aResult,
		int iter, ulong totalFor) {
	ulong patternArray = (blockIdx.x * blockDim.x + threadIdx.x
			+ (totalFor * (iter - 1)));
	if (patternArray < numPatFiltered) {
		ulong pattern = *(patFiltered + patternArray);
		for (ulong row = 0; row < rows; row++) {
			bool bEqual = true;
			for (ulong k = 0; k < cols && bEqual; k++) {
				uint64_t rPattern = *(aResultCols + pattern * cols + k);
				if (((uint64_t) *(mInputData + row * cols + k) & rPattern)
						!= rPattern) {
					bEqual = false;
				}
			}
			if (bEqual) {
				*(aResult + pattern * rows + row) = 1;
			} else {
				*(aResult + pattern * rows + row) = 0;
			}
		}
	}
}

__global__ void generateBiclusters(uint32_t *aResultCols, int id,
		uint32_t *mInputData, long long *patFiltered, uint8_t *aResult,
		int iter, ulong totalFor) {
	ulong patternArray = (blockIdx.x * blockDim.x + threadIdx.x
			+ (totalFor * (iter - 1)));
	if (patternArray < numPatFiltered) {
		ulong pattern = *(patFiltered + patternArray);
		for (ulong row = 0; row < rows; row++) {
			bool bEqual = true;
			for (ulong k = 0; k < cols && bEqual; k++) {
				uint32_t rPattern = *(aResultCols + pattern * cols + k);
				if (((uint32_t) *(mInputData + row * cols + k) & rPattern)
						!= rPattern) {
					bEqual = false;
				}
			}
			if (bEqual) {
				*(aResult + pattern * rows + row) = 1;
			} else {
				*(aResult + pattern * rows + row) = 0;
			}
		}
	}
}

__global__ void generateBiclusters_no_out(uint64_t *aResultCols, int id,
		uint64_t *mInputData, long long *patFiltered, int iter,
		ulong totalFor) {
	ulong patternArray = (blockIdx.x * blockDim.x + threadIdx.x
			+ (totalFor * (iter - 1)));
	if (patternArray < numPatFiltered) {
		ulong pattern = *(patFiltered + patternArray);
		uint64_t numRows = 0;
		for (ulong row = 0; row < rows && numRows <= mnr; row++) {
			bool bEqual = true;
			for (ulong k = 0; k < cols && bEqual; k++) {
				uint64_t rPattern = *(aResultCols + pattern * cols + k);
				if (((uint64_t) *(mInputData + row * cols + k) & rPattern)
						!= rPattern) {
					bEqual = false;
				}
			}
			if (bEqual) {
				numRows++;
			}
		}
		if (numRows >= mnr) {
			atomicAdd(&totalBiclusters, 1);
		}
	}
}

__global__ void generateBiclusters_no_out(uint32_t *aResultCols, int id,
		uint32_t *mInputData, long long *patFiltered, int iter,
		ulong totalFor) {
	ulong patternArray = (blockIdx.x * blockDim.x + threadIdx.x
			+ (totalFor * (iter - 1)));
	if (patternArray < numPatFiltered) {
		ulong pattern = *(patFiltered + patternArray);
		uint32_t numRows = 0;
		for (ulong row = 0; row < rows && numRows < mnr; row++) {
			bool bEqual = true;
			for (ulong k = 0; k < cols && bEqual; k++) {
				uint32_t rPattern = *(aResultCols + pattern * cols + k);
				if (((uint32_t) *(mInputData + row * cols + k) & rPattern)
						!= rPattern) {
					bEqual = false;
				}
			}
			if (bEqual) {
				numRows++;
			}
		}
		if (numRows >= mnr) {
			atomicAdd(&totalBiclusters, 1);
		}
	}
}

// #######
// # CPU #
// #######

// CPU: GENERAL FUNCS
// ------------------

void introduceParameters(char **argv) {

	//PARAMETER AUTOMATIC: pattern Size.
	patternSize = sizeof(void *) * 8;

	// PARAMETER 1: is a one-column file with the names of all the rows, following the order they appear in the dataset. It is used to print information about the elements of final biclusters
	inputFile = (char *) malloc(sizeof(char) * 250);
	inputFile = argv[1];

	// PARAMETER 2: minimum number of rows allowed in a valid bicluster (MNR)
	cMnr = atoi(argv[2]);

	// PARAMETER 3: minimum number of columns allowed in a valid bicluster (MNC)
	cMnc = atoi(argv[3]);

	//PARAMETER 4: OUTPUT
	output = atoi(argv[4]);

	//PARAMETER 5: GPus number
	deviceCount = atoi(argv[5]);

}

void getNumPatterns() {
	maxPatterns = 0;
	for (int i = 0; i < cRows; i++) {
		for (int j = i + 1; j < cRows; j++) {
			maxPatterns++;
		}
	}
}

void prepareGpu1D(ulong lNumber) {
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	lastBlocksGrid = 1;
	maxIteratorGPU = 0;
	maxThreadsPerBlock = lNumber; // Case 1: 0 < lNumber <= prop.maxThreadsPerBlock
	if (lNumber > prop.maxThreadsPerBlock) { // Case 2: lNumber > prop.maxThreadsPerBlock && Supported GPU in a for
		maxThreadsPerBlock = prop.maxThreadsPerBlock;
		maxBlocksPerGrid = lNumber / prop.maxThreadsPerBlock;
		lastBlocksGrid = lNumber / prop.maxThreadsPerBlock;
		if (lNumber % prop.maxThreadsPerBlock != 0) {
			maxBlocksPerGrid++;
			lastBlocksGrid++;
		}
		if (maxBlocksPerGrid > prop.maxGridSize[1]) { // Case 3: Not supported GPU with a for --> Split patterns in multiple for
			maxIteratorGPU = maxBlocksPerGrid / prop.maxGridSize[1];
			lastBlocksGrid = maxBlocksPerGrid
					- (maxIteratorGPU * prop.maxGridSize[1]);
			maxBlocksPerGrid = prop.maxGridSize[1];
		}
	}
}

// CPU: SPECIFIC FUNCS
// -------------------

//String (binary) to UINT64 (64 bits)
uint64_t binaryToDecimal64(const std::string& binary) {
	uint64_t decimal = 0;
	uint64_t p = 1;
	std::string::const_reverse_iterator iter;

	for (iter = binary.rbegin(); iter != binary.rend(); iter++) {
		if (*iter == '1')
			decimal += p;
		p *= 2;
	}

	return decimal;
}

//String (binary) to UINT32 (32 bits)
uint32_t binaryToDecimal32(const std::string& binary) {
	uint32_t decimal = 0;
	uint32_t p = 1;
	std::string::const_reverse_iterator iter;

	for (iter = binary.rbegin(); iter != binary.rend(); iter++) {
		if (*iter == '1')
			decimal += p;
		p *= 2;
	}

	return decimal;
}

//UINT64 to String (binary) (64 bits)
std::string decimalToBinary64(uint64_t decimal) {
	std::string binary;
	while (decimal > 0) {
		if (decimal % 2 == 0)
			binary += '0';
		else
			binary += '1';
		decimal /= 2;
	}
	std::reverse(binary.begin(), binary.end());
	return binary;
}

//UINT32 to String (binary) (32 bits)
std::string decimalToBinary32(uint32_t decimal) {
	std::string binary;
	while (decimal > 0) {
		if (decimal % 2 == 0)
			binary += '0';
		else
			binary += '1';
		decimal /= 2;
	}
	std::reverse(binary.begin(), binary.end());
	return binary;
}

//Read file (64 bits)
uint64_t* fileReader64() {

	// 1) Prepare to GPU: Allocate mArray
	cRows = 0;
	cCols = 0;
	uint64_t *mArray;

	// 2) Prepare ROWS from file
	vector<string> rowsArray_Aux;
	string line;
	ifstream myfile(inputFile);
	if (myfile.is_open()) {
		while (getline(myfile, line)) {
			rowsArray_Aux.push_back(line);
			cRows++;
		}
		myfile.close();

		// 2) Get columns number
		char *cstr_calc_column = strdup(rowsArray_Aux[0].c_str());
		for (int k = strlen(cstr_calc_column) - 1; k >= 0; k--) {
			if (cstr_calc_column[k] == '0' || cstr_calc_column[k] == '1') {
				cCols++;
			}
		}

		cColsTotal = cCols;
		if (cCols % patternSize == 0) {
			cCols = cCols / patternSize;
		} else {
			cCols = (cCols / patternSize) + 1;
		}

		mArray = (uint64_t *) malloc(cRows * cCols * sizeof(uint64_t)); //Store character ASCII

		// 3) Create mArray
		for (int j = 0; j < cRows; j++) {
			char *cstr_rows = strdup(rowsArray_Aux[j].c_str());
			string cWord = "";
			int contmArray = cCols - 1;
			int contcWord = 0;
			for (int k = strlen(cstr_rows) - 1; k >= 0; k--) {
				if (cstr_rows[k] == '0' || cstr_rows[k] == '1') {
					cWord = cstr_rows[k] + cWord;
					contcWord++;
					if (contcWord == patternSize) {
						uint64_t iNumber = binaryToDecimal64(cWord);
						*(mArray + j * cCols + contmArray) = iNumber;
						contcWord = 0;
						cWord = "";
						contmArray--;
					}
				}
			}

			//Last cell
			if (contcWord != 0) {
				uint64_t iNumber = binaryToDecimal64(cWord);
				*(mArray + j * cCols + contmArray) = iNumber;
			}
		}

		for (int i = 0; i < deviceCount; i++) {
			hipSetDevice(i);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&mnc)), &cMnc, sizeof(int), 0,
					hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&mnr)), &cMnr, sizeof(int), 0,
					hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&cols)), &cCols, sizeof(ulong), 0,
					hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&rows)), &cRows, sizeof(ulong), 0,
					hipMemcpyHostToDevice);
		}
	} else {
		cout << "Unable to open file " << endl;
	}

	return mArray;
}

//Read file (32 bits)
uint32_t* fileReader32() {

	// 1) Prepare to GPU: Allocate mArray
	cRows = 0;
	cCols = 0;
	uint32_t *mArray;

	// 2) Prepare ROWS from file
	vector<string> rowsArray_Aux;
	string line;
	ifstream myfile(inputFile);
	if (myfile.is_open()) {
		while (getline(myfile, line)) {
			rowsArray_Aux.push_back(line);
			cRows++;
		}
		myfile.close();

		// 2) Get columns number
		char *cstr_calc_column = strdup(rowsArray_Aux[0].c_str());
		for (int k = strlen(cstr_calc_column) - 1; k >= 0; k--) {
			if (cstr_calc_column[k] == '0' || cstr_calc_column[k] == '1') {
				cCols++;
			}
		}

		cColsTotal = cCols;
		if (cCols % patternSize == 0) {
			cCols = cCols / patternSize;
		} else {
			cCols = (cCols / patternSize) + 1;
		}

		mArray = (uint32_t *) malloc(cRows * cCols * sizeof(uint32_t)); //Store character ASCII

		// 3) Create mArray
		for (int j = 0; j < cRows; j++) {
			char *cstr_rows = strdup(rowsArray_Aux[j].c_str());
			string cWord = "";
			int contmArray = cCols - 1;
			int contcWord = 0;
			for (int k = strlen(cstr_rows) - 1; k >= 0; k--) {
				if (cstr_rows[k] == '0' || cstr_rows[k] == '1') {
					cWord = cstr_rows[k] + cWord;
					contcWord++;
					if (contcWord == patternSize) {
						uint32_t iNumber = binaryToDecimal32(cWord);
						*(mArray + j * cCols + contmArray) = iNumber;
						contcWord = 0;
						cWord = "";
						contmArray--;
					}
				}
			}

			//Last cell
			if (contcWord != 0) {
				uint32_t iNumber = binaryToDecimal32(cWord);
				*(mArray + j * cCols + contmArray) = iNumber;
			}
		}

		for (int i = 0; i < deviceCount; i++) {
			hipSetDevice(i);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&mnc)), &cMnc, sizeof(int), 0,
					hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&mnr)), &cMnr, sizeof(int), 0,
					hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&cols)), &cCols, sizeof(ulong), 0,
					hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(*(&rows)), &cRows, sizeof(ulong), 0,
					hipMemcpyHostToDevice);
		}
	} else {
		cout << "Unable to open file " << endl;
	}

	return mArray;
}

//Get Array Result (64 bits)
uint64_t* getArrayResult64(int id, ulong numPatterns, ulong sizeArray) {
	uint64_t *aResult;
	hipSetDevice(id);
	hipMalloc((void **) &aResult, numPatterns * sizeArray * sizeof(uint64_t));
	hipMemset(aResult, 0, numPatterns * sizeArray * sizeof(uint64_t));
	return aResult;
}

//Get Array Result (32 bits)
uint32_t* getArrayResult32(int id, ulong numPatterns, ulong sizeArray) {
	uint32_t *aResult;
	hipSetDevice(id);
	hipMalloc((void **) &aResult, numPatterns * sizeArray * sizeof(uint32_t));
	hipMemset(aResult, 0, numPatterns * sizeArray * sizeof(uint32_t));
	return aResult;
}

long long* getPatternsFiltered(int id, ulong numPatterns) {
	long long *patFiltered;
	hipMalloc((void **) &patFiltered, numPatterns * sizeof(long long));
	hipMemset(patFiltered, -1, numPatterns * sizeof(long long));
	return patFiltered;
}

unsigned long long int printResults(int id, ulong patternsPerRun,
		uint8_t *aResult, long long *aPatFilteredCpu, uint64_t *aResultColsCpu,
		unsigned long long int cpuNumPatFiltered,
		unsigned long long int totalBic) {
	ulong iTotalRows;
	ofstream myfile;
	string line;
	myfile.open("results_GPU" + to_string(id) + ".txt");
	myfile << "Rows;Cols" << "\n";

	uint8_t *aResultCpu = (uint8_t *) malloc(
			patternsPerRun * cRows * sizeof(uint8_t));
	hipMemcpy(aResultCpu, aResult, patternsPerRun * cRows * sizeof(uint8_t),
			hipMemcpyDeviceToHost);

	for (ulong r = 0; r < cpuNumPatFiltered; r++) {
		iTotalRows = 0;
		line = "";
		long long pattern = *(aPatFilteredCpu + r);
		for (ulong c = 0; c < cRows; c++) {
			if (*(aResultCpu + pattern * cRows + c) == 1) {
				line = line + to_string(c + 1) + ",";
				iTotalRows += 1;
			}
		}
		line.pop_back();
		line = line + ";";

		if (iTotalRows >= cMnr) {
			ulong firstOne = 0, contBit = 0, realPos;
			for (ulong c = 0; c < cCols; c++) {
				uint64_t colsReduce = *(aResultColsCpu + pattern * cCols + c);
				string s = decimalToBinary64(colsReduce);
				firstOne = (cColsTotal - s.length() - (patternSize * c)) + 1;
				for (char& ch : s) {
					if (ch == '1') {
						realPos = firstOne + contBit;
						line = line + to_string(realPos) + ",";
					}
					contBit++;
				}
			}
			line.pop_back();
			myfile << line << "\n";
			totalBic++;
		}
	}
	free(aResultCpu);
	myfile.close();
	return totalBic;
}

unsigned long long int printResults(int id, ulong patternsPerRun,
		uint8_t *aResult, long long *aPatFilteredCpu, uint32_t *aResultColsCpu,
		unsigned long long int cpuNumPatFiltered,
		unsigned long long int totalBic) {
	ulong iTotalRows;
	ofstream myfile;
	string line;
	myfile.open("results_GPU" + to_string(id) + ".txt");
	myfile << "Rows;Cols" << "\n";

	uint8_t *aResultCpu = (uint8_t *) malloc(
			patternsPerRun * cRows * sizeof(uint8_t));
	hipMemcpy(aResultCpu, aResult, patternsPerRun * cRows * sizeof(uint8_t),
			hipMemcpyDeviceToHost);

	for (ulong r = 0; r < cpuNumPatFiltered; r++) {
		iTotalRows = 0;
		line = "";
		long long pattern = *(aPatFilteredCpu + r);
		for (ulong c = 0; c < cRows; c++) {
			if (*(aResultCpu + pattern * cRows + c) == 1) {
				line = line + to_string(c + 1) + ",";
				iTotalRows += 1;
			}
		}
		line.pop_back();
		line = line + ";";

		if (iTotalRows >= cMnr) {
			ulong firstOne = 0, contBit = 0, realPos;
			for (ulong c = 0; c < cCols; c++) {
				uint64_t colsReduce = *(aResultColsCpu + pattern * cCols + c);
				string s = decimalToBinary32(colsReduce);
				firstOne = (cColsTotal - s.length() - (patternSize * c)) + 1;
				for (char& ch : s) {
					if (ch == '1') {
						realPos = firstOne + contBit;
						line = line + to_string(realPos) + ",";
					}
					contBit++;
				}
			}
			line.pop_back();
			myfile << line << "\n";
			totalBic++;
		}
	}
	free(aResultCpu);
	myfile.close();
	return totalBic;
}

void threadsPerDevice_64(int id, hipStream_t s, ulong chunks,
		ulong bicsPerGpuPrevious, ulong patternsPerRun, uint64_t *mInputData,
		mutex *m) {
	hipSetDevice(id);
	ulong totalPatterns = 0;
	unsigned long long int totalBic = 0, totalPatFiltered;

	for (ulong largeScale = 0; largeScale < chunks; largeScale++) {
		uint64_t *aResultCols = getArrayResult64(id, patternsPerRun, cCols);
		long long *patFiltered = getPatternsFiltered(id, patternsPerRun);

		// 1) Generate total patterns
		prepareGpu1D(patternsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++) {
			getPatterns<<<maxBlocksPerGrid, maxThreadsPerBlock,0,s>>>(maxPatterns, aResultCols, id, bicsPerGpuPrevious, mInputData, patFiltered, totalPatterns, patternsPerRun, i,maxThreadsPerBlock*maxBlocksPerGrid);
		}
		getPatterns<<<lastBlocksGrid, maxThreadsPerBlock,0,s>>>(maxPatterns, aResultCols, id, bicsPerGpuPrevious, mInputData, patFiltered, totalPatterns, patternsPerRun, maxIteratorGPU+1, maxThreadsPerBlock*maxBlocksPerGrid);

		unsigned long long int cpuNumPatFiltered;
		hipMemcpyFromSymbol(&cpuNumPatFiltered, HIP_SYMBOL(numPatFiltered),
				sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);

		// 2) Remove duplicate patterns
		uint64_t *aResultColsCpu = (uint64_t *) malloc(
				patternsPerRun * cCols * sizeof(uint64_t));
		hipMemcpy(aResultColsCpu, aResultCols,
				patternsPerRun * cCols * sizeof(uint64_t),
				hipMemcpyDeviceToHost);
		long long *aPatFilteredCpu = (long long *) malloc(
				patternsPerRun * sizeof(long long));
		hipMemcpy(aPatFilteredCpu, patFiltered,
				patternsPerRun * sizeof(long long), hipMemcpyDeviceToHost);
		hipFree(patFiltered);
		std::pair<std::set<vector<uint32_t>>::iterator, bool> ret;
		for (uint64_t i = 0; i < cpuNumPatFiltered; i++) {
			long long pat = *(aPatFilteredCpu + i);
			uint64_t *ptr = &aResultColsCpu[pat * cCols];
			vector<uint64_t> vec(ptr, ptr + cCols);
			vector<uint64_t> vec2 = vec;
			m->lock();
			if (setPatterns64.insert(vec2).second == false) {
				*(aPatFilteredCpu + i) = *(aPatFilteredCpu + cpuNumPatFiltered
						- 1);
				*(aPatFilteredCpu + cpuNumPatFiltered - 1) = -1;
				cpuNumPatFiltered--;
				i--;
			}
			m->unlock();
		}

		hipMalloc((void **) &patFiltered, patternsPerRun * sizeof(long long));
		hipMemcpy(patFiltered, aPatFilteredCpu,
				patternsPerRun * sizeof(long long), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(numPatFiltered), &cpuNumPatFiltered,
				sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);

		// 3)  Generate biclusters
		uint8_t *aResult;
		if (output == 1) {
			hipSetDevice(id);
			hipMalloc((void **) &aResult,
					patternsPerRun * cRows * sizeof(uint8_t));
			hipMemset(aResult, 0, patternsPerRun * cRows * sizeof(uint8_t));
		}

		prepareGpu1D(cpuNumPatFiltered);
		if (output == 1) {
			for (int i = 1; i <= maxIteratorGPU; i++) {
				generateBiclusters<<<maxBlocksPerGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, aResult, i, maxBlocksPerGrid);
			}
			generateBiclusters<<<lastBlocksGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, aResult, maxIteratorGPU+1, maxBlocksPerGrid);
		} else {
			for(int i=1; i <= maxIteratorGPU; i++) {
				generateBiclusters_no_out<<<maxBlocksPerGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, i, maxBlocksPerGrid);
			}
			generateBiclusters_no_out<<<lastBlocksGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, maxIteratorGPU+1, maxBlocksPerGrid);
		}

		// 4) Print results
		if (output == 1) {
			totalBic = printResults(id, patternsPerRun, aResult,
					aPatFilteredCpu, aResultColsCpu, cpuNumPatFiltered,
					totalBic);
		}

		if (output == 0) {
			hipMemcpyFromSymbol(&totalBic, HIP_SYMBOL(totalBiclusters),
					sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);
		}

		hipMemcpyFromSymbol(&totalPatFiltered, HIP_SYMBOL(numPatFiltered),
				sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);
		totales += totalBic;
		totalBic = 0;
		totalPatFiltered = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(totalBiclusters), &totalBic,
				sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(numPatFiltered), &totalPatFiltered,
				sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);
		free(aResultColsCpu);
		free(aPatFilteredCpu);
		hipFree(aResult);
		hipFree(aResultCols);
		hipFree(patFiltered);
		totalPatterns += patternsPerRun;
	}
}

void threadsPerDevice_32(int id, hipStream_t s, ulong chunks,
		ulong bicsPerGpuPrevious, ulong patternsPerRun, uint32_t *mInputData,
		mutex *m) {
	hipSetDevice(id);
	ulong totalPatterns = 0;
	unsigned long long int totalBic = 0, totalPatFiltered;

	for (ulong largeScale = 0; largeScale < chunks; largeScale++) {
		uint32_t *aResultCols = getArrayResult32(id, patternsPerRun, cCols);
		long long *patFiltered = getPatternsFiltered(id, patternsPerRun);

		// 1) Generate total patterns
		prepareGpu1D(patternsPerRun);
		for (int i = 1; i <= maxIteratorGPU; i++) {
			getPatterns<<<maxBlocksPerGrid, maxThreadsPerBlock,0,s>>>(maxPatterns, aResultCols, id, bicsPerGpuPrevious, mInputData, patFiltered, totalPatterns, patternsPerRun, i,maxThreadsPerBlock*maxBlocksPerGrid);
		}
		getPatterns<<<lastBlocksGrid, maxThreadsPerBlock,0,s>>>(maxPatterns, aResultCols, id, bicsPerGpuPrevious, mInputData, patFiltered, totalPatterns, patternsPerRun, maxIteratorGPU+1, maxThreadsPerBlock*maxBlocksPerGrid);

		unsigned long long int cpuNumPatFiltered;
		hipMemcpyFromSymbol(&cpuNumPatFiltered, HIP_SYMBOL(numPatFiltered),
				sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);

		// 2) Remove duplicate patterns
		uint32_t *aResultColsCpu = (uint32_t *) malloc(
				patternsPerRun * cCols * sizeof(uint32_t));
		hipMemcpy(aResultColsCpu, aResultCols,
				patternsPerRun * cCols * sizeof(uint32_t),
				hipMemcpyDeviceToHost);
		long long *aPatFilteredCpu = (long long *) malloc(
				patternsPerRun * sizeof(long long));
		hipMemcpy(aPatFilteredCpu, patFiltered,
				patternsPerRun * sizeof(long long), hipMemcpyDeviceToHost);
		hipFree(patFiltered);
		std::pair<std::set<vector<uint32_t>>::iterator, bool> ret;
		for (uint32_t i = 0; i < cpuNumPatFiltered; i++) {
			long long pat = *(aPatFilteredCpu + i);
			uint32_t *ptr = &aResultColsCpu[pat * cCols];
			vector<uint32_t> vec(ptr, ptr + cCols);
			vector<uint32_t> vec2 = vec;
			m->lock();
			if (setPatterns32.insert(vec2).second == false) {
				*(aPatFilteredCpu + i) = *(aPatFilteredCpu + cpuNumPatFiltered
						- 1);
				*(aPatFilteredCpu + cpuNumPatFiltered - 1) = -1;
				cpuNumPatFiltered--;
				i--;
			}
			m->unlock();
		}
		hipMalloc((void **) &patFiltered, patternsPerRun * sizeof(long long));
		hipMemcpy(patFiltered, aPatFilteredCpu,
				patternsPerRun * sizeof(long long), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(numPatFiltered), &cpuNumPatFiltered,
				sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);

		// 3)  Generate biclusters
		uint8_t *aResult;
		if (output == 1) {
			hipSetDevice(id);
			hipMalloc((void **) &aResult,
					patternsPerRun * cRows * sizeof(uint8_t));
			hipMemset(aResult, 0, patternsPerRun * cRows * sizeof(uint8_t));
		}

		prepareGpu1D(cpuNumPatFiltered);
		if (output == 1) {
			for (int i = 1; i <= maxIteratorGPU; i++) {
				generateBiclusters<<<maxBlocksPerGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, aResult, i, maxBlocksPerGrid);
			}
			generateBiclusters<<<lastBlocksGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, aResult, maxIteratorGPU+1, maxBlocksPerGrid);
		} else {
			for(int i=1; i <= maxIteratorGPU; i++) {
				generateBiclusters_no_out<<<maxBlocksPerGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, i, maxBlocksPerGrid);
			}
			generateBiclusters_no_out<<<lastBlocksGrid, maxThreadsPerBlock,0,s>>>(aResultCols, id, mInputData, patFiltered, maxIteratorGPU+1, maxBlocksPerGrid);
		}

		// 4) Print biclusters
		if (output == 1) {
			totalBic = printResults(id, patternsPerRun, aResult,
					aPatFilteredCpu, aResultColsCpu, cpuNumPatFiltered,
					totalBic);
		}

		if (output == 0) {
			hipMemcpyFromSymbol(&totalBic, HIP_SYMBOL(totalBiclusters),
					sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);
		}

		hipMemcpyFromSymbol(&totalPatFiltered, HIP_SYMBOL(numPatFiltered),
				sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);
		totales += totalBic;
		totalBic = 0;
		totalPatFiltered = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(totalBiclusters), &totalBic,
				sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(numPatFiltered), &totalPatFiltered,
				sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);
		free(aResultColsCpu);
		free(aPatFilteredCpu);
		hipFree(aResult);
		hipFree(aResultCols);
		hipFree(patFiltered);
		totalPatterns += patternsPerRun;
	}
}

void runAlgorithm_64() {

	// 1) Create inputData (Matrix)
	totales = 0;
	uint64_t *mArray = fileReader64();
	getNumPatterns();

	// 2) PREPARING LARGE-SCALE DATA: CHUNKS
	hipStream_t s[deviceCount];
	thread threads[deviceCount];
	ulong chunks[deviceCount], patternsPerRun[deviceCount];
	ulong bicsPerGpu = maxPatterns / deviceCount;
	ulong restBiclustersLastGpu = maxPatterns % deviceCount;

	for (int i = 0; i < deviceCount; i++) {
		hipSetDevice(i);
		hipStreamCreate(&s[i]);
		struct hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		double availableMemory = ((3 * prop.totalGlobalMem) / 4
				- (cRows * cCols * sizeof(uint64_t))); //InputData
		double sizeResult = 0;
		if (output == 1) {
			sizeResult = (bicsPerGpu * cRows * sizeof(char));
		}
		double sizeResultCols = (bicsPerGpu * cCols * sizeof(uint64_t));
		double patFiltered = (bicsPerGpu * sizeof(long long));
		chunks[i] = ((sizeResult + sizeResultCols + patFiltered)
				/ availableMemory) + 1;
		patternsPerRun[i] = bicsPerGpu / chunks[i];
		if (bicsPerGpu % chunks[i] != 0) {
			patternsPerRun[i]++;
		}
		if (deviceCount > 1 && maxPatterns % deviceCount != 0
				&& i == deviceCount - 1) {
			patternsPerRun[i] += restBiclustersLastGpu;
		}
	}

	ulong bicsPerGpuPrevious = 0;
	mutex m;
	for (int i = 0; i < deviceCount; i++) {
		uint64_t *mInputData;
		hipSetDevice(i);
		hipHostMalloc((void**) &mInputData, cRows * cCols * sizeof(uint64_t), hipHostMallocDefault);
		hipMemcpy(mInputData, mArray, cRows * cCols * sizeof(uint64_t),
				hipMemcpyHostToDevice);
		if (i > 0) {
			bicsPerGpuPrevious += chunks[i - 1] * patternsPerRun[i - 1];
		}
		threads[i] = thread(threadsPerDevice_64, i, s[i], chunks[i],
				bicsPerGpuPrevious, patternsPerRun[i], mInputData, &m);
	}

	for (auto& th : threads) {
		th.join();
	}
}

void runAlgorithm_32() {

	// 1) Create inputData (Matrix)
	totales = 0;
	uint32_t *mArray = fileReader32();
	getNumPatterns();

	// 2) PREPARING LARGE-SCALE DATA: CHUNKS
	hipStream_t s[deviceCount];
	thread threads[deviceCount];
	ulong chunks[deviceCount], patternsPerRun[deviceCount];
	ulong bicsPerGpu = maxPatterns / deviceCount;
	ulong restBiclustersLastGpu = maxPatterns % deviceCount;

	for (int i = 0; i < deviceCount; i++) {
		hipSetDevice(i);
		hipStreamCreate(&s[i]);
		struct hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		double availableMemory = ((3 * prop.totalGlobalMem) / 4
				- (cRows * cCols * sizeof(uint32_t))); //InputData
		double sizeResult = 0;
		if (output != 0) {
			sizeResult = (bicsPerGpu * cRows * sizeof(char));
		}
		double sizeResultCols = (bicsPerGpu * cCols * sizeof(uint32_t));
		double patFiltered = (bicsPerGpu * sizeof(long long));
		chunks[i] = ((sizeResult + sizeResultCols + patFiltered)
				/ availableMemory) + 1;
		patternsPerRun[i] = bicsPerGpu / chunks[i];
		if (bicsPerGpu % chunks[i] != 0) {
			patternsPerRun[i]++;
		}
		if (deviceCount > 1 && maxPatterns % deviceCount != 0
				&& i == deviceCount - 1) {
			patternsPerRun[i] += restBiclustersLastGpu;
		}
	}

	ulong bicsPerGpuPrevious = 0;
	mutex m;
	for (int i = 0; i < deviceCount; i++) {
		uint32_t *mInputData;
		hipSetDevice(i);
		hipHostMalloc((void**) &mInputData, cRows * cCols * sizeof(uint32_t), hipHostMallocDefault);
		hipMemcpy(mInputData, mArray, cRows * cCols * sizeof(uint32_t),
				hipMemcpyHostToDevice);
		if (i > 0) {
			bicsPerGpuPrevious += chunks[i - 1] * patternsPerRun[i - 1];
		}
		threads[i] = thread(threadsPerDevice_32, i, s[i], chunks[i],
				bicsPerGpuPrevious, patternsPerRun[i], mInputData, &m);
	}

	for (auto& th : threads) {
		th.join();
	}
}

/*
 ########
 # MAIN #
 ########
 */
int main(int argc, char** argv) {

	introduceParameters(argv);

	//We will work with a maximum size of 32 or 64 bits (patternSize)
	if (patternSize == 64) {
		runAlgorithm_64();
	} else {
		runAlgorithm_32();
	}

	// PRINT BITPAT
	printf("Resume:\n========================\n");
	printf("Dataset filename: %s\n", inputFile);
	printf("Dataset size (rows,columns): %d, %d\n", cRows, cCols);
	printf("GPUs devices: %d\n", deviceCount);
	printf("Pattern size: %lu\n", patternSize);
	printf("MNC value: %lu\n", cMnc);
	printf("MNR value: %lu\n", cMnr);

	printf("\nResults:\n========================\n");
	cout << "Biclusters: " << totales << endl;

	return hipDeviceReset();
}
